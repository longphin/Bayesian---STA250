#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{

__global__ void
rtruncnorm_kernel(float *x, int n,
		  float *mu, float *sigma,
	   	  float *a, float *b,
		  int len_mu, int len_sigma,
		  int len_a, int len_b,
		  int maxRejections,
		  int rng_c)
{
	int rng_a=1; // These can easily be made as argument, but I'm lazy.
	int rng_b=2;

	// Usual block/thread indexing...
	int myblock = blockIdx.x + blockIdx.y * gridDim.x;
	int blocksize = blockDim.x * blockDim.y * blockDim.z;
	int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	int idx = myblock * blocksize + subthread;

	// Determine indexes for vectors if using recycling.
	// Note: A good programmer would avoid the code repitions.
	int ind_mu = ((len_mu<2) ? 0 : (idx % len_mu));
	int ind_sigma = ((len_sigma<2) ? 0 : (idx % len_sigma));
	int ind_a = ((len_a<2) ? 0 : (idx % len_a));
	int ind_b = ((len_b<2) ? 0 : (idx % len_b));

	if (idx < n){ 
		// Set up RNG
		hiprandState rng;
		hiprand_init(rng_a+idx*rng_b, rng_c, 0, &rng);

		// Sample truncated normal, doing rejection-sampling
		for(int i=0; i<maxRejections; i++)
		{
			float samp=mu[ind_mu]+sigma[ind_sigma]*hiprand_normal(&rng);
			if(a[ind_a]<=samp && samp<=b[ind_b])
			{
				x[idx]=samp;
				return;
			}
		}
		// Could not sample using rejection-sampling.
		// Simply sample from Uniform(a,b).
		x[idx]=hiprand_uniform(&rng)*(a[ind_a]-b[ind_b])+a[ind_a];
	}
	return;
}

} // END extern "C"

