#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{

	__global__ void
		rtruncnorm_kernel(float *vals, int n,
				float *mu, float *sigma,
				float *lo, float *hi,
				int mu_len, int sigma_len,
				int lo_len, int hi_len,
				int maxRejections,
				int rng_a,
				int rng_b,
				int rng_c)
		{
			// Usual block/thread indexing...
			int myblock = blockIdx.x + blockIdx.y * gridDim.x;
			int blocksize = blockDim.x * blockDim.y * blockDim.z;
			int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
			int idx = myblock * blocksize + subthread;
			if (idx < N){ 
				// Set up RNG
				hiprandState rng;
				hiprand_init(rng_a+idx*rng_b, rng_c, 0, &rng);

				// Sample truncated normal, doing rejection-sampling
				accepted=0 // 0 indicates no acceptance yet
					for(int i=0; i<maxRejections; i++)
					{
						float samp=mu[idx]+sigma[idx]*hiprand_normal(&rng);
						if(a[idx]<=samp && samp<=b[idx])
						{
							x[idx]=samp;
							break;
						}else{
							if(i==maxRejections-1)
							{
								// Could not sample using rejection-sampling. Let's cheat.
								x[idx]=hiprand_uniform(&rng)*(a[idx]-b[idx])+a[idx]; // sample x~Uniform(a,b) 
							}
						}
					}

				return;
			}

		} // END extern "C"

